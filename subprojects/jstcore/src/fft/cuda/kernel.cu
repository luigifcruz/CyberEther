#include "hip/hip_runtime.h"
#include "jstcore/fft/cuda/kernel.hpp"

namespace Jetstream {
namespace FFT {
namespace Kernel {

static __device__ inline float clamp(const float x, const float a, float b) {
    return (x < a) ? a : (b < x) ? b : x;
}

static __device__ inline float scale(const float x, const float min, const float max) {
    return (x - min) / (max - min);
}

static __device__ inline float amplt(const hipFloatComplex x, const int n) {
    return 20 * log10(hipCabsf(x) / n);
}

static __global__ void pre(hipfftComplex* c, const hipfftComplex* win, const uint n){
    const int numThreads = blockDim.x * gridDim.x;
    const int threadID = blockIdx.x * blockDim.x + threadIdx.x;

    for (int i = threadID; i < n; i += numThreads) {
        c[i] = hipCmulf(c[i], win[i]);
    }
}

static __global__ void post(const hipfftComplex* c, float* r,
    const float min, const float max, const uint n){
    const int numThreads = blockDim.x * gridDim.x;
    const int threadID = blockIdx.x * blockDim.x + threadIdx.x;

    float tmp;
    for (int i = threadID; i < n; i += numThreads) {
        tmp = amplt(c[i], n);
        tmp = scale(tmp, min, max);
        tmp = clamp(tmp, 0.0f, 1.0f);

        r[i] = tmp;
    }
}

void PreFFT(const int grid, const int block, const hipStream_t hip_stream,
        hipfftComplex* fft, const hipfftComplex* win_in, const int len) {
    pre<<<grid, block, 0, hip_stream>>>(fft, win_in, len);
}

void PostFFT(const int grid, const int block, const hipStream_t hip_stream,
        const hipfftComplex* fft_in, float* fft_out, const float min, const float max, const int len) {
    post<<<grid, block, 0, hip_stream>>>(fft_in, fft_out, min, max, len);
}

} // namespace Kernel
} // namespace FFT
} // namespace Jetstream
