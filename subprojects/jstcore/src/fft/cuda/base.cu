#include "hip/hip_runtime.h"
#include "jstcore/fft/cuda.hpp"

namespace Jetstream::FFT {

static __device__ inline float clamp(const float x, const float a, float b) {
    return (x < a) ? a : (b < x) ? b : x;
}

static __device__ inline float scale(const float x, const float min, const float max) {
    return (x - min) / (max - min);
}

static __device__ inline float amplt(const hipFloatComplex x, const int n) {
    return 20 * log10(hipCabsf(x) / n);
}

static __global__ void pre(hipfftComplex* c, const hipfftComplex* win, const uint n){
    const int numThreads = blockDim.x * gridDim.x;
    const int threadID = blockIdx.x * blockDim.x + threadIdx.x;

    for (int i = threadID; i < n; i += numThreads) {
        c[i] = hipCmulf(c[i], win[i]);
    }
}

static __global__ void post(const hipfftComplex* c, float* r,
    const float min, const float max, const uint n){
    const int numThreads = blockDim.x * gridDim.x;
    const int threadID = blockIdx.x * blockDim.x + threadIdx.x;

    float tmp;
    for (int i = threadID; i < n; i += numThreads) {
        tmp = amplt(c[i], n);
        tmp = scale(tmp, min, max);
        tmp = clamp(tmp, 0.0f, 1.0f);

        r[i] = tmp;
    }
}

CUDA::CUDA(const Config & config, const Input & input) : Generic(config, input) {
    CUDA_CHECK_THROW(hipStreamCreateWithFlags(&stream, hipStreamNonBlocking));
    CUDA_CHECK_THROW(hipHostRegister(input.in.buf.data(), input.in.buf.size() * sizeof(input.in.buf[0]),
            hipHostRegisterReadOnly));

    fft_len = input.in.buf.size() * sizeof(input.in.buf[0]);
    CUDA_CHECK_THROW(hipMalloc(&fft_dptr, fft_len));

    win_len = input.in.buf.size() * sizeof(input.in.buf[0]);
    CUDA_CHECK_THROW(hipMalloc(&win_dptr, win_len));
    CUDA_CHECK_THROW(hipMemcpy(win_dptr, window.data(), win_len, hipMemcpyHostToDevice));

    out_len = input.in.buf.size() * sizeof(float);
    CUDA_CHECK_THROW(hipMallocManaged(&out_dptr, out_len));
    out.location = Locale::CUDA; // | Locale::CPU;
    out.buf = VF32{out_dptr, input.in.buf.size()};

    hipfftPlan1d(&plan, input.in.buf.size(), HIPFFT_C2C, 1);
    hipfftSetStream(plan, stream);
}

FFT::CUDA::~CUDA() {
    hipHostUnregister(input.in.buf.data());
    hipfftDestroy(plan);
    hipFree(fft_dptr);
    hipFree(out_dptr);
    hipStreamDestroy(stream);
}

Result CUDA::underlyingCompute() {
    int N = input.in.buf.size();
    int threads = 32;
    int blocks = (N + threads - 1) / threads;
    auto [min, max] = config.amplitude;

    CUDA_CHECK(hipMemcpyAsync(fft_dptr, input.in.buf.data(), fft_len, hipMemcpyHostToDevice, stream));
    pre<<<blocks, threads, 0, stream>>>(fft_dptr, win_dptr, N);
    hipfftExecC2C(plan, fft_dptr, fft_dptr, HIPFFT_FORWARD);
    post<<<blocks, threads, 0, stream>>>(fft_dptr, out_dptr, min, max, N);
    CUDA_CHECK(hipStreamSynchronize(stream));

    return Result::SUCCESS;
}

} // namespace Jetstream::FFT
