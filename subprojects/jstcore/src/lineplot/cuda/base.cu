#include "jstcore/lineplot/cuda.hpp"

namespace Jetstream::Lineplot {

CUDA::CUDA(const Config & config, const Input & input) : Generic(config, input) {
    plot_len = plot.size() * sizeof(plot[0]);
    JST_CUDA_CHECK_THROW(hipStreamCreateWithFlags(&stream, hipStreamNonBlocking));
    JST_CUDA_CHECK_THROW(hipMalloc(&plot_dptr, plot_len));
    JST_CUDA_CHECK_THROW(hipMemcpy(plot_dptr, plot.data(), plot_len, hipMemcpyHostToDevice));
    JST_CHECK_THROW(this->initRender(plot_dptr, config.render->cudaInteropSupported()));
}

CUDA::~CUDA() {
    hipFree(plot_dptr);
    hipStreamDestroy(stream);
}

Result CUDA::underlyingCompute() {
    size_t elementSize = sizeof(float);
    size_t srcPitchInBytes = 1 * elementSize;
    size_t dstPitchInBytes = 3 * elementSize;
    size_t width = 1 * elementSize;
    size_t height = input.in.buf.size();

    JST_CUDA_CHECK(hipMemcpy2DAsync(plot_dptr + 1, dstPitchInBytes, input.in.buf.data(), srcPitchInBytes,
        width, height, hipMemcpyDeviceToDevice, stream));
    JST_CUDA_CHECK(hipStreamSynchronize(stream));

    return Result::SUCCESS;
}

} // namespace Jetstream::Lineplot
