#include "jstcore/lineplot/cuda.hpp"

namespace Jetstream::Lineplot {

CUDA::CUDA(const Config & config, const Input & input) : Generic(config, input) {
    plot_len = plot.size() * sizeof(plot[0]);
    CUDA_CHECK_THROW(hipStreamCreateWithFlags(&stream, hipStreamNonBlocking));
    CUDA_CHECK_THROW(hipMalloc(&plot_dptr, plot_len));
    CUDA_CHECK_THROW(hipMemcpy(plot_dptr, plot.data(), plot_len, hipMemcpyHostToDevice));
    JETSTREAM_CHECK_THROW(this->initRender(plot_dptr, config.render->cudaInteropSupported()));
}

CUDA::~CUDA() {
    hipFree(plot_dptr);
    hipStreamDestroy(stream);
}

Result CUDA::underlyingCompute() {
    size_t elementSize = sizeof(float);
    size_t srcPitchInBytes = 1 * elementSize;
    size_t dstPitchInBytes = 3 * elementSize;
    size_t width = 1 * elementSize;
    size_t height = input.in.buf.size();

    CUDA_CHECK(hipMemcpy2DAsync(plot_dptr + 1, dstPitchInBytes, in.buf.data(), srcPitchInBytes,
        width, height, hipMemcpyDeviceToDevice, stream));
    CUDA_CHECK(hipStreamSynchronize(stream));

    return Result::SUCCESS;
}

} // namespace Jetstream::Lineplot
