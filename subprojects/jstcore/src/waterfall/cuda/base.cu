#include "jstcore/waterfall/cuda.hpp"

namespace Jetstream::Waterfall {

CUDA::CUDA(const Config & config, const Input & input) : Generic(config, input) {
    ymax = config.size.height;
    CUDA_CHECK_THROW(hipStreamCreateWithFlags(&stream, hipStreamNonBlocking));
    CUDA_CHECK_THROW(hipMalloc(&out_dptr, in.buf.size() * ymax * sizeof(float)));
    JETSTREAM_CHECK_THROW(this->initRender((uint8_t*)out_dptr, config.render->cudaInteropSupported()));
}

CUDA::~CUDA() {
    hipFree(out_dptr);
    hipStreamDestroy(stream);
}

Result CUDA::underlyingCompute() {
    CUDA_CHECK(hipMemcpyAsync(out_dptr+(inc*input.in.buf.size()), input.in.buf.data(),
            sizeof(float)*input.in.buf.size(), hipMemcpyDeviceToDevice, stream));
    CUDA_CHECK(hipStreamSynchronize(stream));
    return Result::SUCCESS;
}

} // namespace Jetstream::Waterfall
