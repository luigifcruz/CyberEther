#include "jstcore/waterfall/cuda.hpp"

namespace Jetstream::Waterfall {

CUDA::CUDA(const Config & config, const Input & input) : Generic(config, input) {
    ymax = config.size.height;
    JST_CUDA_CHECK_THROW(hipStreamCreateWithFlags(&stream, hipStreamNonBlocking));
    JST_CUDA_CHECK_THROW(hipMalloc(&out_dptr, input.in.buf.size() * ymax * sizeof(float)));
    JST_CHECK_THROW(this->initRender((uint8_t*)out_dptr, config.render->cudaInteropSupported()));
}

CUDA::~CUDA() {
    hipFree(out_dptr);
    hipStreamDestroy(stream);
}

Result CUDA::underlyingCompute() {
    JST_CUDA_CHECK(hipMemcpyAsync(out_dptr+(inc*input.in.buf.size()), input.in.buf.data(),
            sizeof(float)*input.in.buf.size(), hipMemcpyDeviceToDevice, stream));
    JST_CUDA_CHECK(hipStreamSynchronize(stream));
    return Result::SUCCESS;
}

} // namespace Jetstream::Waterfall
