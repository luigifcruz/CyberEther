#include "jetstream/modules/lineplot/cuda.hpp"

namespace Jetstream {

Lineplot::CUDA::CUDA(const Config & c, Manifest & i) : Lineplot(c, i) {
    plot_len = plot.size() * sizeof(plot[0]);
    CUDA_CHECK_THROW(hipStreamCreateWithFlags(&stream, hipStreamNonBlocking));
    CUDA_CHECK_THROW(hipMalloc(&plot_dptr, plot_len));
    CUDA_CHECK_THROW(hipMemcpy(plot_dptr, plot.data(), plot_len, hipMemcpyHostToDevice));
    JETSTREAM_CHECK_THROW(this->_initRender(plot_dptr, cfg.render->cudaInteropSupported()));
}

Lineplot::CUDA::~CUDA() {
    hipFree(plot_dptr);
    hipStreamDestroy(stream);
}

Result Lineplot::CUDA::_compute() {
    size_t elementSize = sizeof(float);
    size_t srcPitchInBytes = 1 * elementSize;
    size_t dstPitchInBytes = 3 * elementSize;
    size_t width = 1 * elementSize;
    size_t height = in.buf.size();

    CUDA_CHECK(hipMemcpy2DAsync(plot_dptr + 1, dstPitchInBytes, in.buf.data(), srcPitchInBytes,
        width, height, hipMemcpyDeviceToDevice, stream));
    CUDA_CHECK(hipStreamSynchronize(stream));

    return Result::SUCCESS;
}

Result Lineplot::CUDA::_present() {
    lineVertex->update();

    return Result::SUCCESS;
}

} // namespace Jetstream
