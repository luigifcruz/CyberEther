#include "hip/hip_runtime.h"
#include "jetstream/modules/fft/cuda.hpp"

namespace Jetstream {

static __device__ inline float clamp(const float x, const float a, float b) {
    return (x < a) ? a : (b < x) ? b : x;
}

static __device__ inline float scale(const float x, const float min, const float max) {
    return (x - min) / (max - min);
}

static __device__ inline float amplt(const hipFloatComplex x, const int n) {
    return 20 * log10(hipCabsf(x) / n);
}

static __global__ void pre(hipfftComplex* c, const hipfftComplex* win, const uint n){
    const int numThreads = blockDim.x * gridDim.x;
    const int threadID = blockIdx.x * blockDim.x + threadIdx.x;

    for (int i = threadID; i < n; i += numThreads) {
        c[i] = hipCmulf(c[i], win[i]);
    }
}

static __global__ void post(const hipfftComplex* c, float* r,
    const float min, const float max, const uint n){
    const int numThreads = blockDim.x * gridDim.x;
    const int threadID = blockIdx.x * blockDim.x + threadIdx.x;

    float tmp;
    for (int i = threadID; i < n; i += numThreads) {
        tmp = amplt(c[i], n);
        tmp = scale(tmp, min, max);
        tmp = clamp(tmp, 0.0f, 1.0f);

        r[i] = tmp;
    }
}

FFT::CUDA::CUDA(const Config & c, Manifest & i) : FFT(c, i) {
    CUDA_CHECK_THROW(hipStreamCreateWithFlags(&stream, hipStreamNonBlocking));
    CUDA_CHECK_THROW(hipHostRegister(in.buf.data(), in.buf.size() * sizeof(in.buf[0]),
            hipHostRegisterReadOnly));

    fft_len = in.buf.size() * sizeof(in.buf[0]);
    CUDA_CHECK_THROW(hipMalloc(&fft_dptr, fft_len));

    win_len = in.buf.size() * sizeof(in.buf[0]);
    CUDA_CHECK_THROW(hipMalloc(&win_dptr, win_len));
    CUDA_CHECK_THROW(hipMemcpy(win_dptr, window.data(), win_len, hipMemcpyHostToDevice));

    out_len = in.buf.size() * sizeof(float);
    CUDA_CHECK_THROW(hipMallocManaged(&out_dptr, out_len));
    out.buf = nonstd::span<float>{out_dptr, in.buf.size()};

    hipfftPlan1d(&plan, in.buf.size(), HIPFFT_C2C, 1);
    hipfftSetStream(plan, stream);

    out_manifest["output0"] = out;
}

FFT::CUDA::~CUDA() {
    hipHostUnregister(in.buf.data());
    hipfftDestroy(plan);
    hipFree(fft_dptr);
    hipFree(out_dptr);
    hipStreamDestroy(stream);
}

Result FFT::CUDA::compute() {
    DEBUG_PUSH("compute_fft");

    int N = in.buf.size();
    int threads = 32;
    int blocks = (N + threads - 1) / threads;
    auto [min, max] = cfg.amplitude;

    CUDA_CHECK(hipMemcpyAsync(fft_dptr, in.buf.data(), fft_len, hipMemcpyHostToDevice, stream));
    pre<<<blocks, threads, 0, stream>>>(fft_dptr, win_dptr, N);
    hipfftExecC2C(plan, fft_dptr, fft_dptr, HIPFFT_FORWARD);
    post<<<blocks, threads, 0, stream>>>(fft_dptr, out_dptr, min, max, N);
    CUDA_CHECK(hipStreamSynchronize(stream));

    DEBUG_POP();
    return Result::SUCCESS;
}

Result FFT::CUDA::present() {
    return Result::SUCCESS;
}

} // namespace Jetstream
