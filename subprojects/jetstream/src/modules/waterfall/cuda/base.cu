#include "jetstream/modules/waterfall/cuda.hpp"

namespace Jetstream {

Waterfall::CUDA::CUDA(const Config& cfg, IO & input) : Waterfall(cfg, input) {
    ymax = cfg.size.height;
    CUDA_CHECK_THROW(hipStreamCreateWithFlags(&stream, hipStreamNonBlocking));
    CUDA_CHECK_THROW(hipMalloc(&out_dptr, in.buf.size() * ymax * sizeof(float)));
    JETSTREAM_CHECK_THROW(this->_initRender((uint8_t*)out_dptr, cfg.render->cudaInteropSupported()));
}

Waterfall::CUDA::~CUDA() {
    hipFree(out_dptr);
    hipStreamDestroy(stream);
}

Result Waterfall::CUDA::_compute() {
    CUDA_CHECK(hipMemcpyAsync(out_dptr+(inc*in.buf.size()), in.buf.data(), sizeof(float)*in.buf.size(),
            hipMemcpyDeviceToDevice, stream));
    CUDA_CHECK(hipStreamSynchronize(stream));

    return Result::SUCCESS;
}

} // namespace Jetstream
